#include "hip/hip_runtime.h"
/*
* Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <stdio.h>
extern "C" {
    #include "cuda_sort.h"
}

////////////////////////////////////////////////////////////////////////////////
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
////////////////////////////////////////////////////////////////////////////////
__device__ void selection_sort( int *data, int left, int right )
{
  for( int i = left ; i <= right ; ++i ){
    int min_val = data[i];
    int min_idx = i;

    // Find the smallest value in the range [left, right].
    for( int j = i+1 ; j <= right ; ++j ){
      int val_j = data[j];
      if( val_j < min_val ){
        min_idx = j;
        min_val = val_j;
      }
    }

    // Swap the values.
    if( i != min_idx ){
      data[min_idx] = data[i];
      data[i] = min_val;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_simple_quicksort(int *data, int left, int right, int depth ){
    //If we're too deep or there are few elements left, we use an insertion sort...
    if( depth >= MAX_DEPTH || right-left <= INSERTION_SORT ){
        selection_sort( data, left, right );
        return;
    }

    hipStream_t s,s1;
    int *lptr = data+left;
    int *rptr = data+right;
    int  pivot = data[(left+right)/2];

    int lval;
    int rval;

    int nright, nleft;

    // Do the partitioning.
    while (lptr <= rptr){
        // Find the next left- and right-hand values to swap
        lval = *lptr;
        rval = *rptr;

        // Move the left pointer as long as the pointed element is smaller than the pivot.
        while (lval < pivot && lptr < data+right){
            lptr++;
            lval = *lptr;
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot && rptr > data+left){
            rptr--;
            rval = *rptr;
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr){
            *lptr = rval;
            *rptr = lval;
            lptr++;
            rptr--;
        }
    }

    // Now the recursive part
    nright = rptr - data;
    nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data)){
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right){
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Call the quicksort kernel from the host.
////////////////////////////////////////////////////////////////////////////////
extern "C"
void gpu_qsort(int *data, int n){
    int* gpuData;
    int left = 0;
    int right = n-1;

    // Prepare CDP for the max depth 'MAX_DEPTH'.
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);

    // Allocate GPU memory.
    hipMalloc((void**)&gpuData,n*sizeof(int));
    hipMemcpy(gpuData,data, n*sizeof(int), hipMemcpyHostToDevice);

    // Launch on device
    cdp_simple_quicksort<<< 1, 1 >>>(gpuData, left, right, 0);
    hipDeviceSynchronize();

    // Copy back
    hipMemcpy(data,gpuData, n*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(gpuData);
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
}
